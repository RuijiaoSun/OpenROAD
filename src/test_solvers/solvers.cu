#include <hip/hip_runtime.h>
#include <cusp/krylov/gmres.h>
#include <cusp/monitor.h>
#include <cusp/print.h>

#include <iostream>

#include "solvers.h"
// where to perform the computation
typedef cusp::device_memory MemorySpace;

std::vector<float> cudaSolver(const std::vector<int>* rowIndex,
                              const std::vector<int>* columnIndex,
                              const std::vector<float>* value,
                              const std::vector<float>* b)
{
  std::vector<float> x;
  x.resize(b->size());

  // Set handler
  hipsolverSpHandle_t handleCusolver = NULL;
  hipsparseHandle_t handleCusparse = NULL;
  hipStream_t stream = NULL;

  // Initialize handler
  CUSOLVER_ERROR(hipsolverSpCreate(&handleCusolver));
  CUSPARSE_ERROR(hipsparseCreate(&handleCusparse));
  CUDA_ERROR(hipStreamCreate(&stream));
  CUSOLVER_ERROR(hipsolverSpSetStream(handleCusolver, stream));
  CUSPARSE_ERROR(hipsparseSetStream(handleCusparse, stream));

  int *d_rowIndex, *d_columnIndex;
  float *d_value, *d_b, *d_x;  // d_p is some mediate vector
  int nnz = rowIndex->size();  // Number of non-zero values in A
  int m = b->size();           // Rows of the matrx A
  float tol = 1.e-10;
  int reorder = 0;
  int singularity = 0;

  // Allocate space on device
  CUDA_ERROR(hipMalloc((void**) &d_rowIndex, sizeof(int) * nnz));
  CUDA_ERROR(hipMalloc((void**) &d_columnIndex, sizeof(int) * nnz));
  CUDA_ERROR(hipMalloc((void**) &d_value, sizeof(float) * nnz));
  CUDA_ERROR(hipMalloc((void**) &d_b, sizeof(float) * m));
  CUDA_ERROR(hipMalloc((void**) &d_x, sizeof(float) * m));

  // Copy data (COO storage method)
  CUDA_ERROR(hipMemcpy(
      d_rowIndex, rowIndex->data(), sizeof(int) * nnz, hipMemcpyHostToDevice));
  CUDA_ERROR(hipMemcpy(d_columnIndex,
                        columnIndex->data(),
                        sizeof(int) * nnz,
                        hipMemcpyHostToDevice));
  CUDA_ERROR(hipMemcpy(
      d_value, value->data(), sizeof(float) * nnz, hipMemcpyHostToDevice));
  CUDA_ERROR(
      hipMemcpy(d_b, b->data(), sizeof(float) * m, hipMemcpyHostToDevice));
  CUDA_ERROR(
      hipMemcpy(d_x, x.data(), sizeof(float) * m, hipMemcpyHostToDevice));

  // Create and define cusparse descriptor
  hipsparseMatDescr_t descrA = NULL;
  CUSPARSE_ERROR(hipsparseCreateMatDescr(&descrA));
  CUSPARSE_ERROR(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CUSPARSE_ERROR(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));

  // COO to CSR
  int* d_csrRowIndex = NULL;
  CUDA_ERROR(hipMalloc((void**) &d_csrRowIndex,
                        sizeof(int) * (m + 1)));  // Array length: (m+1)
  CUSPARSE_ERROR(hipsparseXcoo2csr(handleCusparse,
                                  d_rowIndex,
                                  nnz,
                                  m,
                                  d_csrRowIndex,
                                  HIPSPARSE_INDEX_BASE_ZERO));

  // QR method
  // https://docs.nvidia.com/cuda/cusolver/index.html
  CUSOLVER_ERROR(hipsolverSpScsrlsvqr(handleCusolver,
                                     m,
                                     nnz,
                                     descrA,
                                     d_value,
                                     d_csrRowIndex,
                                     d_columnIndex,
                                     d_b,
                                     tol,
                                     reorder,
                                     d_x,
                                     &singularity));

  // Copy data
  CUDA_ERROR(hipMemcpyAsync(
      x.data(), d_x, sizeof(float) * m, hipMemcpyDeviceToHost, stream));

  // Release space on the GPU
  CUDA_ERROR(hipFree(d_rowIndex));
  CUDA_ERROR(hipFree(d_columnIndex));
  CUDA_ERROR(hipFree(d_value));
  CUDA_ERROR(hipFree(d_b));
  CUDA_ERROR(hipFree(d_x));
  CUDA_ERROR(hipFree(d_csrRowIndex));
  return x;
}

std::vector<float> cuspSolver(const std::vector<int>* rowIndex,
                              const std::vector<int>* columnIndex,
                              const std::vector<float>* value,
                              const std::vector<float>* b)
{
  std::vector<float> x = {1, 1};

  int *d_rowIndex, *d_columnIndex;
  float *d_value, *d_b, *d_x;  // d_p is some mediate vector
  int nnz = rowIndex->size();  // Number of non-zero values in A
  int m = b->size();           // Rows of the matrx A

  // Allocate space on device
  CUDA_ERROR(hipMalloc((void**) &d_rowIndex, sizeof(int) * nnz));
  CUDA_ERROR(hipMalloc((void**) &d_columnIndex, sizeof(int) * nnz));
  CUDA_ERROR(hipMalloc((void**) &d_value, sizeof(float) * nnz));
  CUDA_ERROR(hipMalloc((void**) &d_b, sizeof(float) * m));
  CUDA_ERROR(hipMalloc((void**) &d_x, sizeof(float) * m));

  // Copy data (COO storage method)
  CUDA_ERROR(hipMemcpy(
      d_rowIndex, rowIndex->data(), sizeof(int) * nnz, hipMemcpyHostToDevice));
  CUDA_ERROR(hipMemcpy(d_columnIndex,
                        columnIndex->data(),
                        sizeof(int) * nnz,
                        hipMemcpyHostToDevice));
  CUDA_ERROR(hipMemcpy(
      d_value, value->data(), sizeof(float) * nnz, hipMemcpyHostToDevice));
  CUDA_ERROR(
      hipMemcpy(d_b, b->data(), sizeof(float) * m, hipMemcpyHostToDevice));
  CUDA_ERROR(
      hipMemcpy(d_x, x.data(), sizeof(float) * m, hipMemcpyHostToDevice));
  thrust::device_ptr<int> p_rowInd(d_rowIndex);
  thrust::device_ptr<int> p_colInd(d_columnIndex);
  thrust::device_ptr<float> p_val(d_value);
  thrust::device_ptr<float> p_x(d_x);
  thrust::device_ptr<float> p_b(d_b);

  // use array1d_view to wrap the individual arrays
  typedef typename cusp::array1d_view<thrust::device_ptr<int>>
      DeviceIndexArrayView;
  typedef typename cusp::array1d_view<thrust::device_ptr<float>>
      DeviceValueArrayView;
  DeviceIndexArrayView row_indices(p_rowInd, p_rowInd + nnz);
  DeviceIndexArrayView column_indices(p_colInd, p_colInd + nnz);
  DeviceValueArrayView values(p_val, p_val + nnz);
  DeviceValueArrayView b_x(p_x, p_x + m);
  DeviceValueArrayView b_b(p_b, p_b + m);

  // combine the three array1d_views into a coo_matrix_view
  typedef cusp::coo_matrix_view<DeviceIndexArrayView,
                                DeviceIndexArrayView,
                                DeviceValueArrayView>
      DeviceView;

  // construct a coo_matrix_view from the array1d_views
  DeviceView b_A(m, m, nnz, row_indices, column_indices, values);

  // set stopping criteria:
  //  iteration_limit    = 100
  //  relative_tolerance = 1e-5
  //  absolute_tolerance = 0
  //  verbose            = true
  cusp::monitor<float> monitor(b_b, 100, 1e-10, 0, true);

  // solve the linear system A * x = b with the Conjugate Gradient method
  cusp::krylov::gmres(b_A, b_x, b_b, 50, monitor);
  cusp::print(b_A);
  cusp::print(b_x);
  cusp::print(b_b);

  // Copy data
  CUDA_ERROR(hipMemcpyAsync(
      x.data(), d_x, sizeof(float) * m, hipMemcpyDeviceToHost));

  // Release space on the GPU
  CUDA_ERROR(hipFree(d_rowIndex));
  CUDA_ERROR(hipFree(d_columnIndex));
  CUDA_ERROR(hipFree(d_value));
  CUDA_ERROR(hipFree(d_b));
  CUDA_ERROR(hipFree(d_x));
  return x;
}
